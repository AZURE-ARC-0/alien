#include "hip/hip_runtime.h"
﻿#include "RenderingKernelsLauncher.cuh"

#include "RenderingData.cuh"
#include "RenderingKernels.cuh"

void _RenderingKernelsLauncher::drawImage(
    GpuSettings const& gpuSettings,
    float2 rectUpperLeft,
    float2 rectLowerRight,
    int2 imageSize,
    float zoom,
    SimulationData data,
    RenderingData renderingData)
{
    uint64_t* targetImage = renderingData.imageData;

    KERNEL_CALL(cudaDrawBackground, targetImage, imageSize, data.worldSize, zoom, rectUpperLeft, rectLowerRight);
    KERNEL_CALL(cudaDrawCells, data.timestep, data.worldSize, rectUpperLeft, rectLowerRight, data.objects.cellPointers, targetImage, imageSize, zoom);
    KERNEL_CALL(cudaDrawParticles, data.worldSize, rectUpperLeft, rectLowerRight, data.objects.particlePointers, targetImage, imageSize, zoom);
    KERNEL_CALL_1_1(cudaDrawRadiationSources, targetImage, rectUpperLeft, data.worldSize, imageSize, zoom);
    KERNEL_CALL(cudaDrawRepetition, data.worldSize, imageSize, rectUpperLeft, rectLowerRight, targetImage, zoom);
}
